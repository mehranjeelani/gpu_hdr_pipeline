
#include <hip/hip_runtime.h>
#include <cstdint>
#include "stdio.h"
#include "math.h"

__global__ void tonemap_kernel(std::uint32_t* out, const float* in, int width, int height, float exposure)
{
	
	int x = (blockIdx.x*blockDim.x + threadIdx.x)*4;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
//	if(x==0 and y==0)
//		printf("Hello CUda\n");
	if(y< height && x <= width*4-4){
	float input_tone[3] = {in[y*width*4+x] * exposure,in[y*4*width+x+1]*exposure,in[y*4*width+x+2]*exposure};
	float output_tone[3];
	output_tone[0] = input_tone[0]*(0.9036*input_tone[0] + 0.018)/(input_tone[0]*(0.8748*input_tone[0]+0.354)+0.14);
	output_tone[1] = input_tone[1]*(0.9036*input_tone[1] + 0.018)/(input_tone[1]*(0.8748*input_tone[1]+0.354)+0.14);
	output_tone[2] = input_tone[2]*(0.9036*input_tone[2] + 0.018)/(input_tone[2]*(0.8748*input_tone[2]+0.354)+0.14);
	
	output_tone[0] = (output_tone[0] <= 0.0031308 ? 12.92 * output_tone[0] : 1.055 * powf(output_tone[0],1/2.4) - 0.055);
	output_tone[1] = (output_tone[1] <= 0.0031308 ? 12.92 * output_tone[1] : 1.055 * powf(output_tone[1],1/2.4) - 0.055);
	output_tone[2] = (output_tone[2] <= 0.0031308 ? 12.92 * output_tone[2] : 1.055 * powf(output_tone[2],1/2.4) - 0.055);

	output_tone[0] = (output_tone[0]>1 ? 255 : rintf(255*output_tone[0]));
	output_tone[1] = (output_tone[1]>1? 255 : rintf(255*output_tone[1]));
	output_tone[2] = (output_tone[2]>1? 255 : rintf(255*output_tone[2]));

	out[y*width+x/4] = 0 |(std::uint8_t)(output_tone[2]) <<16 | (std::uint8_t)(output_tone[1]) <<8 | (std::uint8_t)(output_tone[0]);

	/*
	output_tone[0] = compute(in[y*width*4+x]*exposure,width,height);
	output_tone[1] = compute(in[y*width*4+x+1] * exposure,width,height);
	output_tone[2] = compute(in[y*width*4+x+2] * exposure,width,height);
	//std::uint8_t scaled_output[3];
	*/
	/*
	for(int i=0;i<3;i++){
		output_tone[i] = input_tone[i]*(0.9036*input_tone[i] + 0.018)/(input_tone[i]*(0.8748*input_tone[i]+0.354)+0.14);
		if(output_tone[i] <= 0.0031308)
			output_tone[i] = 12.92 * output_tone[i];
		else
			output_tone[i] = 1.055 * powf(output_tone[i],1/2.4) - 0.055;
		output_tone[i] = (output_tone[i]>1) ? 255 : rintf(255*output_tone[i]);
			
	}
	*/
	//out[y*width+x/4] = 0 |(std::uint8_t)(output_tone[2]) <<16 | (std::uint8_t)(output_tone[1]) <<8 | (std::uint8_t)(output_tone[0]);
	/*
	out[y*width+x/4] = (std::uint8_t)(output_tone[2]) <<16;
	out[y*width+x/4] =  (std::uint8_t)(output_tone[1])<<8;
	out[y*width+x/4] =  (std::uint8_t)(output_tone[0]);
//	float output_tone = input_tone*(0.9036*input_tone + 0.018)/(input_tone*(0.8748*input_tone+0.354)+0.14);
//	float output_gamma;
//	if(output_tone <= 0.0031308)
//		output_gamma = 12.92 * output_tone;
//	else
//		output_gamma = 1.055 * pow(output_tone,1/2.4) - 0.055;
//	std::uint8_t scaled_value = (output_gamma>1) ? 255 : 255*output_gamma;
	*/
		
	}
	
	
}


void tonemap(std::uint32_t* out, const float* in, int width, int height, float exposure, float brightpass_threshold)
{	dim3 blockSize (32,32,1);
	dim3 gridSize (width/blockSize.x+1,height/blockSize.y+1,1);
//	printf("before tonemapkernel\n");	
	tonemap_kernel<<<gridSize, blockSize>>>(out, in, width, height, exposure);
	hipDeviceSynchronize();
//	printf("after tonemapkernel\n");
}

